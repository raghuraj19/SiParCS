#include "hip/hip_runtime.h"
#include <stdio.h>
#include <geninputdata.h>
#include <timer.h>
#include <bdcuda.h>

#include "hip/hip_runtime.h"
#include "" 

__global__ void odefun_cuda(real *d_Wx, real *d_Wy, real *d_GammaHV, int *d_neighbors, int size, int stencil_size, real *d_u, real *d_v, real *d_temp_sol, real *d_step_sol)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int j, index;
	real val = 0, x_sol = 0, y_sol = 0, hv_sol = 0;
	real temp_dWx, temp_dWy, temp_dGammaHV;
	real temp_du, temp_dv;

	if (tid < size)
	{
		for (j = 0; j < stencil_size; j++)
		{
			temp_dWx = d_Wx[j * stencil_size + tid];
			temp_dWy = d_Wy[j * stencil_size + tid];
			temp_dGammaHV = d_GammaHV[j * stencil_size + tid];
			index = d_neighbors[j * stencil_size + tid] - 1;
			val = d_temp_sol[index];
			x_sol = x_sol + temp_dWx * val;
			y_sol = y_sol + temp_dWy * val;
			hv_sol = hv_sol + temp_dGammaHV * val;
		}
		
		temp_du = d_u[tid];
		temp_dv = d_v[tid];
	
		d_step_sol[tid] = (-1 * temp_du * x_sol) + (-1 * temp_dv * y_sol) + hv_sol;
	}	
}


void odefun_gpu(float time_step, real *temp_sol, real *step_sol)
{
	int i, j;

	//INITIALIZE CUDA EVENTS
	hipEvent_t start,stop;
	float elapsedTime;

	//CREATING EVENTS
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/* pack data for gpu */

	real *d_Wx, *d_Wy, *d_GammaHV, *d_temp_sol, *d_step_sol, *d_u, *d_v;
	real *h_Wx, *h_Wy, *h_GammaHV;
	int *h_neighbors, *d_neighbors;

	h_Wx = (real *)malloc(nodes->internal_count * stencil_size * sizeof(real));
	h_Wy = (real *)malloc(nodes->internal_count * stencil_size * sizeof(real));
	h_GammaHV = (real *)malloc(nodes->internal_count * stencil_size * sizeof(real));
	h_neighbors = (int *)malloc(nodes->internal_count * stencil_size * sizeof(int));


	for (i = 0; i < nodes->internal_count; i++)
        {
                for (j = 0; j < stencil_size; j++)
                {
                        h_Wx[nodes->internal_count * j + i] = Wx->val[i][j];
                        h_Wy[nodes->internal_count * j + i] = Wy->val[i][j];
                        h_GammaHV[nodes->internal_count * j + i] = GammaHV[i][j];
                        h_neighbors[nodes->internal_count * j + i] = nodes->neighbors[i][j];
                }
        }

	hipMalloc(&d_Wx, nodes->internal_count * stencil_size * sizeof(real));
	hipMalloc(&d_Wy, nodes->internal_count * stencil_size * sizeof(real));
	hipMalloc(&d_GammaHV, nodes->internal_count * stencil_size * sizeof(real));
	hipMalloc(&d_neighbors, nodes->internal_count * stencil_size * sizeof(int));
	hipMalloc(&d_u, nodes->total_count * sizeof(real));
	hipMalloc(&d_v, nodes->total_count * sizeof(real));
	hipMalloc(&d_temp_sol, nodes->total_count * sizeof(real));
	hipMalloc(&d_step_sol, nodes->internal_count * sizeof(real));

	hipMemcpy(d_Wx, h_Wx, nodes->internal_count * stencil_size * sizeof(real), hipMemcpyHostToDevice);
	hipMemcpy(d_Wy, h_Wy, nodes->internal_count * stencil_size * sizeof(real), hipMemcpyHostToDevice);
	hipMemcpy(d_GammaHV, h_GammaHV, nodes->internal_count * stencil_size * sizeof(real), hipMemcpyHostToDevice);
	hipMemcpy(d_neighbors, h_neighbors, nodes->internal_count * stencil_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_u, nodes->u, nodes->total_count * sizeof(real), hipMemcpyHostToDevice);
	hipMemcpy(d_v, nodes->v, nodes->total_count * sizeof(real), hipMemcpyHostToDevice);
	hipMemcpy(d_temp_sol, temp_sol, nodes->total_count * sizeof(real), hipMemcpyHostToDevice);

	int threads_per_block = 256;
	hipEventRecord(start,0);
	odefun_cuda<<<((nodes->internal_count - 1) / threads_per_block) + 1, threads_per_block>>>(d_Wx, d_Wy, d_GammaHV, d_neighbors, nodes->internal_count, stencil_size, d_u, d_v, d_temp_sol, d_step_sol);
	hipDeviceSynchronize();
	
	//FINISH RECORDING
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	//CALCULATE ELAPSED TIME
	hipEventElapsedTime(&elapsedTime,start,stop);

	//DISPLAY COMPUTATION TIME
	//printf("Elapsed Time = %f\n",elapsedTime);
	
	hipEventRecord(start,0);
	hipMemcpy(step_sol, d_step_sol, nodes->internal_count * sizeof(real), hipMemcpyDeviceToHost);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime,start,stop);
        printf("Elapsed Time = %f\n",elapsedTime);
	/* unpack data from gpu */

	hipFree(d_Wx);
	hipFree(d_Wy);
	hipFree(d_GammaHV);
	hipFree(d_temp_sol);
	hipFree(d_step_sol);
	hipFree(d_u);
	hipFree(d_v);

	free(h_Wx);
	free(h_Wy);
	free(h_GammaHV);
	free(h_neighbors);

	return;
}
